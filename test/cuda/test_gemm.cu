#include <hiprand.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include "../../include/blas_cuda.hpp"
#include "../../include/_utils_cuda.hpp"

#if defined(USE_DOUBLE)
    #define data_t double
    #define GEMM cuda::dgemm
    #define GEMM_CUBLAS hipblasDgemm
    #define RANDOM hiprandGenerateNormalDouble
    #define ALLCLOSE cuda::dscaledAllclose
    #define TEST_CASE CUDA_DGEMM_TEST
    #define TESTER CUDA_DGEMM_TESTER
#else
    #define data_t float
    #define GEMM cuda::sgemm
    #define GEMM_CUBLAS hipblasSgemm
    #define RANDOM hiprandGenerateNormal
    #define ALLCLOSE cuda::sscaledAllclose
    #define TEST_CASE CUDA_SGEMM_TEST
    #define TESTER CUDA_SGEMM_TESTER
#endif


bool TESTER(int M, int N, int K) {

    hiprandGenerator_t gen;

    data_t *A, *B, *C, *cublas_C;
    CUDA_CALL(hipMalloc((void**)&A, N*K*sizeof(data_t)));
    CUDA_CALL(hipMalloc((void**)&B, K*M*sizeof(data_t)));
    CUDA_CALL(hipMalloc((void**)&C, N*M*sizeof(data_t)));
    CUDA_CALL(hipMalloc((void**)&cublas_C, N*M*sizeof(data_t)));

    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    CURAND_CALL(RANDOM(gen, A, N*K, 0.0, 1.0));
    CURAND_CALL(RANDOM(gen, B, K*M, 0.0, 1.0));

    GEMM(A, B, C, N);

    data_t alpha = 1.0f;
    data_t beta = 0.0f;
    hipblasHandle_t handle; 
    hipblasCreate(&handle);
    
    GEMM_CUBLAS(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        N, N, N,
        &alpha,
        B, N,
        A, N,
        &beta,
        cublas_C, N);
    hipDeviceSynchronize();
        
    bool is_passed = ALLCLOSE(C, cublas_C, N*M);

    CURAND_CALL(hiprandDestroyGenerator(gen));
    hipblasDestroy(handle);
    CUDA_CALL(hipFree(A));
    CUDA_CALL(hipFree(B));
    CUDA_CALL(hipFree(C));
    CUDA_CALL(hipFree(cublas_C));

    return is_passed;
}

// NOTE: fp32/1024, the error does not fit into the defaults rtol and atol

TEST(TEST_CASE, same_size_matrixes_1) {
    int M = 2048, N = 2048, K = 2048;
    ASSERT_TRUE(TESTER(M, N, K));
}

TEST(TEST_CASE, same_size_matrixes_2) {
    int M = 4096, N = 4096, K = 4096;
    ASSERT_TRUE(TESTER(M, N, K));
}

TEST(TEST_CASE, same_size_matrixes_3) {
    int M = 8192, N = 8192, K = 8192;
    ASSERT_TRUE(TESTER(M, N, K));
}