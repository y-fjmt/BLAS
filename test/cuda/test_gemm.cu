#include <hiprand.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include "../../include/blas_cuda.hpp"
#include "../../include/_utils_cuda.hpp"

#if defined(USE_DOUBLE)
    #define data_t double
    #define GEMM cuda::dgemm
    #define GEMM_CUBLAS hipblasDgemm
    #define RANDOM hiprandGenerateUniformDouble
    #define ALLCLOSE cuda::dallclose
    #define TEST_CASE CUDA_DGEMM_TEST
    #define TESTER CUDA_DGEMM_TESTER
#else
    #define data_t float
    #define GEMM cuda::sgemm
    #define GEMM_CUBLAS hipblasSgemm
    #define RANDOM hiprandGenerateUniform
    #define ALLCLOSE cuda::sallclose
    #define TEST_CASE CUDA_SGEMM_TEST
    #define TESTER CUDA_SGEMM_TESTER
#endif


bool TESTER(int M, int N, int K) {

    hiprandGenerator_t gen;

    data_t *A, *B, *C, *cublas_C;
    CUDA_CALL(hipMalloc((void**)&A, N*K*sizeof(data_t)));
    CUDA_CALL(hipMalloc((void**)&B, K*M*sizeof(data_t)));
    CUDA_CALL(hipMalloc((void**)&C, N*M*sizeof(data_t)));
    CUDA_CALL(hipMalloc((void**)&cublas_C, N*M*sizeof(data_t)));

    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    CURAND_CALL(RANDOM(gen, A, N*K));
    CURAND_CALL(RANDOM(gen, B, K*M));
    CURAND_CALL(RANDOM(gen, C, N*M));
    CURAND_CALL(RANDOM(gen, cublas_C, N*M));
    
    GEMM(A, B, C, N);

    data_t alpha = 1.0f;
    data_t beta = 0.0f;
    hipblasHandle_t handle; 
    hipblasCreate(&handle);

    GEMM_CUBLAS(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            N, N, N,
            &alpha,
            A, N,
            B, N,
            &beta,
            cublas_C, N);

    hipDeviceSynchronize();
    bool is_passed = ALLCLOSE(C, cublas_C, N*M);


    hipDeviceSynchronize();

    CURAND_CALL(hiprandDestroyGenerator(gen));
    hipblasDestroy(handle);
    CUDA_CALL(hipFree(A));
    CUDA_CALL(hipFree(B));
    CUDA_CALL(hipFree(C));
    CUDA_CALL(hipFree(cublas_C));

    return is_passed;
}


TEST(TEST_CASE, same_size_matrixes) {
    int M = 1024, N = 1024, L = 1024;
    ASSERT_TRUE(TESTER(M, N, L));
}