#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include "../include/utils.hpp"

using namespace std;

#define data_t float

#if defined(data_t) && data_t == float
    #define GEMM_NAME cuda_sgemm
    #define GEMM_KERNEL_NAME cuda_sgemm_kernel
#elif defined(data_t) && data_t == double
    #define GEMM_NAME cuda_dgemm
    #define GEMM_KERNEL_NAME cuda_dgemm_Kernel
#else
    #error data_t is not given or invalid data type.
#endif


__global__ void GEMM_KERNEL_NAME(data_t *A, data_t *B, data_t *C, size_t N) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= N or j >= N) return;

    C[i*N+j] = 0.0;
    for (int k = 0; k < N; k++) {
        C[i*N+j] += A[i*N+k] * B[k*N+j];
    }

}


void GEMM_NAME(data_t *A, data_t *B, data_t *C, size_t N, 
                float *kernel_time=nullptr) {

    data_t *d_A, *d_B, *d_C;
    CHECK(hipMalloc((void**)&d_A, N*N*sizeof(data_t)));
    CHECK(hipMalloc((void**)&d_B, N*N*sizeof(data_t)));
    CHECK(hipMalloc((void**)&d_C, N*N*sizeof(data_t)));

    CHECK(hipMemcpy(d_A, A, N*N*sizeof(data_t), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, B, N*N*sizeof(data_t), hipMemcpyHostToDevice));

    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));

    int tpb = 16;  /* sqrt(1024) */
    dim3 ThreadsPerBlocks(tpb, tpb);
    dim3 BlocksPerGrids((N + tpb - 1) / tpb, (N + tpb - 1) / tpb);

    CHECK(hipEventRecord(start));
    GEMM_KERNEL_NAME<<<BlocksPerGrids, ThreadsPerBlocks>>>(d_A, d_B, d_C, N);

    CHECK(hipEventRecord(end));
    CHECK(hipEventSynchronize(end));

    CHECK(hipMemcpy(C, d_C, N*N*sizeof(data_t), hipMemcpyDeviceToHost));

    if (kernel_time) {
        hipEventElapsedTime(kernel_time, start, end);
    }
    
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(end));

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
}