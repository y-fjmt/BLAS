#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>

#if defined(data_t) && data_t == float
    #define GEMM_NAME cuda_sgemm
    #define GEMM_KERNEL_NAME cuda_sgemm_kernel
#elif defined(data_t) && data_t == double
    #define GEMM_NAME cuda_dgemm
    #define GEMM_KERNEL_NAME cuda_dgemm_Kernel
#else
    #error data_t is not given or invalid data type.
#endif


__global__ void GEMM_KERNEL_NAME(data_t *A, data_t *B, data_t *C, size_t N) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= N or j >= N) return;

    C[i*N+j] = 0.0;
    for (int k = 0; k < N; k++) {
        C[i*N+j] += A[i*N+k] * B[k*N+j];
    }

    if (i == 0 and j == 0) {
        printf("%f\n", C[i*N+j]);
    }

}


void GEMM_NAME(data_t *A, data_t *B, data_t *C, size_t N, 
                float *kernel_time=nullptr) {

    data_t *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N*N*sizeof(data_t));
    hipMalloc((void**)&d_B, N*N*sizeof(data_t));
    hipMalloc((void**)&d_C, N*N*sizeof(data_t));

    hipMemcpy(d_A, A, N*N*sizeof(data_t), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*N*sizeof(data_t), hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    int threads_1d = 1024;
    int blocks_1d = (N + threads_1d - 1) / threads_1d;
    dim3 threads(threads_1d, threads_1d);
    dim3 blocks(blocks_1d, blocks_1d);

    hipEventRecord(start);
    GEMM_KERNEL_NAME<<<blocks, threads>>>(d_A, d_B, d_C, N);
    hipEventRecord(end);
    hipEventSynchronize(end);
    hipDeviceSynchronize();

    hipMemcpy(d_C, C, N*N*sizeof(data_t), hipMemcpyDeviceToHost);

    if (kernel_time) {
        hipEventElapsedTime(kernel_time, start, end);
    }
    
    hipEventDestroy(start);
    hipEventDestroy(end);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}