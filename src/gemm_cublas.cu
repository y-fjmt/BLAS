#include <hipblas.h>
#include <hip/hip_runtime.h>

#if defined(data_t) && data_t == float
    #define CUBLAS_GEMM_FN hipblasSgemm
    #define CUBLAS_GEMM cuda_blas_sgemm
#elif defined(data_t) && data_t == double
    #define CUBLAS_GEMM_FN hipblasDgemm
    #define CUBLAS_GEMM cuda_blas_dgemm
#else
    #error data_t is not given or invalid data type.
#endif


void CUBLAS_GEMM(data_t* A, data_t* B, data_t* C, size_t N,
            float *kernel_time=nullptr) {

    data_t *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N*N*sizeof(data_t));
    hipMalloc((void**)&d_B, N*N*sizeof(data_t));
    hipMalloc((void**)&d_C, N*N*sizeof(data_t));
    hipMemcpy(d_A, A, N*N*sizeof(data_t), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*N*sizeof(data_t), hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipblasHandle_t handle;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipblasCreate(&handle);

    const data_t alpha = 1.0;
    const data_t beta = 0.0;

    hipEventRecord(start);

    CUBLAS_GEMM_FN(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        N, N, N,
        &alpha,
        A, N,
        B, N,
        &beta,
        C, N
    );

    hipEventRecord(end);
    hipEventSynchronize(end);

    hipMemcpy(d_C, C, N*N*sizeof(data_t), hipMemcpyDeviceToHost);

    if (kernel_time) {
        hipEventElapsedTime(kernel_time, start, end);
    }

    hipEventDestroy(start);
    hipEventDestroy(end);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}