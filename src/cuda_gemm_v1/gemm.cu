#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>
#include "../../include/utils.hpp"

using namespace std;

#if defined(USE_DOUBLE)
    #define data_t double
    #define GEMM_NAME cuda_dgemm
    #define GEMM_KERNEL_NAME cuda_dgemm_Kernel
#else
    #define data_t float
    #define GEMM_NAME cuda_sgemm
    #define GEMM_KERNEL_NAME cuda_sgemm_kernel
#endif

__global__ void GEMM_KERNEL_NAME(data_t *A, data_t *B, data_t *C, size_t N) {

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= N or j >= N) return;

    data_t c = 0.0;
    for (int k = 0; k < N; k++) {
        c += A[i*N+k] * B[k*N+j];
    }
    C[i*N+j] = c;

}


void GEMM_NAME(data_t *A, data_t *B, data_t *C, size_t N, 
                float *kernel_time=nullptr) {

    data_t *d_A, *d_B, *d_C;
    CHECK(hipMalloc((void**)&d_A, N*N*sizeof(data_t)));
    CHECK(hipMalloc((void**)&d_B, N*N*sizeof(data_t)));
    CHECK(hipMalloc((void**)&d_C, N*N*sizeof(data_t)));

    CHECK(hipMemcpy(d_A, A, N*N*sizeof(data_t), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, B, N*N*sizeof(data_t), hipMemcpyHostToDevice));

    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));

    dim3 ThreadsPerBlocks(32, 32);
    dim3 BlocksPerGrids((N + 32 - 1) / 32,
                        (N + 32 - 1) / 32);

    CHECK(hipEventRecord(start));
    GEMM_KERNEL_NAME<<<BlocksPerGrids, ThreadsPerBlocks>>>(d_A, d_B, d_C, N);

    CHECK(hipEventRecord(end));
    CHECK(hipEventSynchronize(end));

    CHECK(hipMemcpy(C, d_C, N*N*sizeof(data_t), hipMemcpyDeviceToHost));

    if (kernel_time) {
        CHECK(hipEventElapsedTime(kernel_time, start, end));
    }
    
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(end));

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
}