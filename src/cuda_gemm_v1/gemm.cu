#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>
#include "../../include/utils.hpp"

using namespace std;

#if defined(USE_DOUBLE)
    #define data_t double
    #define GEMM_NAME cuda_dgemm
    #define GEMM_KERNEL_NAME cuda_dgemm_Kernel
#else
    #define data_t float
    #define GEMM_NAME cuda_sgemm
    #define GEMM_KERNEL_NAME cuda_sgemm_kernel
#endif

#define BLOCK 32

__global__ void GEMM_KERNEL_NAME(data_t *A, data_t *B, data_t *C, size_t N) {

    const int i = blockIdx.x * BLOCK + (threadIdx.x / BLOCK);
    const int j = blockIdx.y * BLOCK + (threadIdx.x % BLOCK);
    if (i >= N or j >= N) return;

    C[i*N+j] = 0.0;
    for (int k = 0; k < N; k++) {
        C[i*N+j] += A[i*N+k] * B[k*N+j];
    }

    // int y_global = blockIdx.y * TILE_SIZE + threadIdx.y; 
    // int x_global = blockIdx.x * TILE_SIZE + threadIdx.x; 

    // __shared__ data_t local_A[TILE_SIZE][TILE_SIZE];
    // __shared__ data_t local_B[TILE_SIZE][TILE_SIZE];

    // // load into shared memory
    // local_A[threadIdx.y][threadIdx.x] = A[y_global*N+x_global];
    // local_B[threadIdx.y][threadIdx.x] = B[y_global*N+x_global];

    // __syncthreads();

}


void GEMM_NAME(data_t *A, data_t *B, data_t *C, size_t N, 
                float *kernel_time=nullptr) {

    data_t *d_A, *d_B, *d_C;
    CHECK(hipMalloc((void**)&d_A, N*N*sizeof(data_t)));
    CHECK(hipMalloc((void**)&d_B, N*N*sizeof(data_t)));
    CHECK(hipMalloc((void**)&d_C, N*N*sizeof(data_t)));

    CHECK(hipMemcpy(d_A, A, N*N*sizeof(data_t), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, B, N*N*sizeof(data_t), hipMemcpyHostToDevice));

    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));

    dim3 ThreadsPerBlocks(BLOCK*BLOCK);
    dim3 BlocksPerGrids((N + BLOCK - 1) / BLOCK,
                        (N + BLOCK - 1) / BLOCK);

    CHECK(hipEventRecord(start));
    GEMM_KERNEL_NAME<<<BlocksPerGrids, ThreadsPerBlocks>>>(d_A, d_B, d_C, N);

    CHECK(hipEventRecord(end));
    CHECK(hipEventSynchronize(end));

    CHECK(hipMemcpy(C, d_C, N*N*sizeof(data_t), hipMemcpyDeviceToHost));

    if (kernel_time) {
        CHECK(hipEventElapsedTime(kernel_time, start, end));
    }
    
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(end));

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
}