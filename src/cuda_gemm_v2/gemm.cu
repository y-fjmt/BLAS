#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>
#include "../../include/utils.hpp"

using namespace std;

#if defined(USE_DOUBLE)
    #define data_t double
    #define GEMM_NAME cuda_dgemm
    #define GEMM_KERNEL_NAME cuda_dgemm_Kernel
#else
    #define data_t float
    #define GEMM_NAME cuda_sgemm
    #define GEMM_KERNEL_NAME cuda_sgemm_kernel
#endif

#define TS_M 32
#define TS_N 32
#define TS_L 32

__global__ void GEMM_KERNEL_NAME(data_t *A, data_t *B, data_t *C, size_t N) {

    int M = N;
    int L = N;

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= M || j >= N) return;

    __shared__ data_t _A[TS_M][TS_L];
    __shared__ data_t _B[TS_L][TS_N];
    
    data_t c = 0.0;

    i = threadIdx.y;
    j = threadIdx.x;

    int I = blockIdx.y * TS_M;
    int J = blockIdx.x * TS_N;

    for (int K = 0; K < N; K+=TS_L) {
        
        // load into shared memory        
        // FIXME: (K+j),(K+i)は正方行列限定
        _A[i][j] = A[(I+i)*N + (K+j)];
        _B[i][j] = B[(K+i)*N + (J+j)];

        // Coalescing
        // _B[i][j] = B[(J+j)*N]+(K+i);

        __syncthreads();

        // compute sub matrix
        for (int k = 0; k < TS_L; k++) {
            c += _A[i][k] * _B[k][j];
        }

        __syncthreads();
    }

    C[(I+i) * N + (J+j)] = c;

}


void GEMM_NAME(data_t *A, data_t *B, data_t *C, size_t N, 
                float *kernel_time=nullptr) {

    data_t *d_A, *d_B, *d_C;
    CHECK(hipMalloc((void**)&d_A, N*N*sizeof(data_t)));
    CHECK(hipMalloc((void**)&d_B, N*N*sizeof(data_t)));
    CHECK(hipMalloc((void**)&d_C, N*N*sizeof(data_t)));

    CHECK(hipMemcpy(d_A, A, N*N*sizeof(data_t), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, B, N*N*sizeof(data_t), hipMemcpyHostToDevice));

    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));

    dim3 ThreadsPerBlocks(TS_M, TS_N);
    dim3 BlocksPerGrids((N + TS_M - 1) / TS_M,
                        (N + TS_N - 1) / TS_N);

    CHECK(hipEventRecord(start));
    GEMM_KERNEL_NAME<<<BlocksPerGrids, ThreadsPerBlocks>>>(d_A, d_B, d_C, N);

    CHECK(hipEventRecord(end));
    CHECK(hipEventSynchronize(end));

    CHECK(hipMemcpy(C, d_C, N*N*sizeof(data_t), hipMemcpyDeviceToHost));

    if (kernel_time) {
        CHECK(hipEventElapsedTime(kernel_time, start, end));
    }
    
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(end));

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
}