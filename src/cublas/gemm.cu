#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../../include/utils.hpp"

#if defined(USE_DOUBLE)
    #define data_t double
    #define GEMM_NAME cublas_dgemm
    #define CUBLAS_GEMM_NAME hipblasDgemm
#else
    #define data_t float
    #define GEMM_NAME cublas_sgemm
    #define CUBLAS_GEMM_NAME hipblasSgemm
#endif

void GEMM_NAME(data_t *A, data_t *B, data_t *C, size_t N, 
    float *kernel_time=nullptr) {

    data_t *d_A, *d_B, *d_C;
    CHECK(hipMalloc((void**)&d_A, N*N*sizeof(data_t)));
    CHECK(hipMalloc((void**)&d_B, N*N*sizeof(data_t)));
    CHECK(hipMalloc((void**)&d_C, N*N*sizeof(data_t)));

    CHECK(hipMemcpy(d_A, A, N*N*sizeof(data_t), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, B, N*N*sizeof(data_t), hipMemcpyHostToDevice));

    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));

    const data_t alpha = 1.0f, beta = 0.0f;

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    CHECK(hipEventRecord(start));

    CHECK_CUBLAS(
        CUBLAS_GEMM_NAME(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            N, N, N,
            &alpha,
            d_B, N,
            d_A, N,
            &beta,
            d_C, N)
    );

    CHECK(hipEventRecord(end));
    CHECK(hipEventSynchronize(end));

    hipDeviceSynchronize();

    CHECK(hipMemcpy(C, d_C, N*N*sizeof(data_t), hipMemcpyDeviceToHost));

    if (kernel_time) {
        CHECK(hipEventElapsedTime(kernel_time, start, end));
    }

    CHECK_CUBLAS(hipblasDestroy(handle));
    
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(end));

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
}