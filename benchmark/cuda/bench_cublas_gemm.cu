#include <hipblas.h>
#include <hiprand.h>
#include <benchmark/benchmark.h>
#include "../../include/blas_cuda.hpp"
#include "../../include/_utils_cuda.hpp"

#if defined(USE_DOUBLE)
    #define data_t double
    #define GEMM_CUBLAS hipblasDgemm
    #define RANDOM hiprandGenerateNormalDouble
    #define BM_GEMM bench_dgemm_cublas
    #define BM_NAME "CUBLAS_DGEMM"
#else
    #define data_t float
    #define GEMM_CUBLAS hipblasSgemm
    #define RANDOM hiprandGenerateNormal
    #define BM_GEMM bench_sgemm_cublas
    #define BM_NAME "CUBLAS_SGEMM"
#endif

static void BM_GEMM(benchmark::State& state) {
    
    // Setup (not mesured)
    int M = state.range(0);
    int N = state.range(1);
    int K = state.range(2);

    hiprandGenerator_t gen;

    data_t *A, *B, *C;
    CUDA_CALL(hipMalloc((void**)&A, N*K*sizeof(data_t)));
    CUDA_CALL(hipMalloc((void**)&B, K*M*sizeof(data_t)));
    CUDA_CALL(hipMalloc((void**)&C, N*M*sizeof(data_t)));

    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    CURAND_CALL(RANDOM(gen, A, N*K, 0.0, 1.0));
    CURAND_CALL(RANDOM(gen, B, K*M, 0.0, 1.0));

    data_t alpha = 1.0f;
    data_t beta = 0.0f;
    hipblasHandle_t handle; 
    hipblasCreate(&handle);

    // benchmark
    for (auto _ : state) {
        GEMM_CUBLAS(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            N, N, N,
            &alpha,
            B, N,
            A, N,
            &beta,
            C, N);
        hipDeviceSynchronize();
        benchmark::DoNotOptimize(C);
    }

    // Teardown (not mesured)
    double gflop = 2.0 * M * N * K * state.iterations() / 1e9;
    state.counters["GFLOPS"] = benchmark::Counter(gflop, benchmark::Counter::kIsRate);

    CURAND_CALL(hiprandDestroyGenerator(gen));
    hipblasDestroy(handle);
    CUDA_CALL(hipFree(A));
    CUDA_CALL(hipFree(B));
    CUDA_CALL(hipFree(C));
}

// register benchmarks
BENCHMARK(BM_GEMM)->Name(BM_NAME)->Args({1024, 1024, 1024});
BENCHMARK(BM_GEMM)->Name(BM_NAME)->Args({2048, 2048, 2048});
BENCHMARK(BM_GEMM)->Name(BM_NAME)->Args({4096, 4096, 4096});
BENCHMARK(BM_GEMM)->Name(BM_NAME)->Args({8192, 8192, 8192});
BENCHMARK(BM_GEMM)->Name(BM_NAME)->Args({16384, 16384, 16384});
